#include "hip/hip_runtime.h"
/*  Copyright (c) 2019
 *
 *  This program is free software: you can redistribute it and/or modify it
 *  under the terms of the GNU General Public License as published by the
 *  Free Software Foundation, either version 3 of the License, or (at your
 *  option) any later version.
 *
 *  This program is distributed in the hope that it will be useful, but
 *  WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 *  Authors:
 *      Simone Rossi <simone.rossi@eurecom.fr>
 *      Maurizio Filippone <maurizio.filippone@eurecom.fr>
 */


#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


// ELEMENTARY_LOG2SIZE can be changed to another positive integer.
#define ELEMENTARY_LOG2SIZE 11


/*
Single in-global memory FWHT pass.
For strides exceeding ELEMENTARY_LOG2SIZE.
*/
template <typename scalar_t>
__global__ void fwht_batch2_kernel(scalar_t* __restrict__ d_output, int stride) {
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int N   = blockDim.x * gridDim.x * 4;

    scalar_t *d_Src = d_output  + blockIdx.y * N;
    scalar_t *d_Dst = d_output + blockIdx.y * N;

    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    scalar_t D0 = d_Src[i0];
    scalar_t D1 = d_Src[i1];
    scalar_t D2 = d_Src[i2];
    scalar_t D3 = d_Src[i3];

    scalar_t T;
    T = D0;
    D0        = D0 + D2;
    D2        = T - D2;
    T = D1;
    D1        = D1 + D3;
    D3        = T - D3;
    T = D0;
    d_Dst[i0] = D0 + D1;
    d_Dst[i1] = T - D1;
    T = D2;
    d_Dst[i2] = D2 + D3;
    d_Dst[i3] = T - D3;
}


/*
Elementary in-shared memory FWHT.
For strides below (or equal to) ELEMENTARY_LOG2SIZE.
*/
template <typename scalar_t>
__global__ void fwht_batch1_kernel(scalar_t* __restrict__ d_output, int log2d)
{
    const int N = 1 << log2d;
    const int base = blockIdx.x << log2d;

    extern __shared__ unsigned char shared_mem[];
    scalar_t *s_data = reinterpret_cast<scalar_t *>(shared_mem);

    scalar_t *d_Src = d_output + base;
    scalar_t *d_Dst = d_output + base;

    for (int pos = threadIdx.x; pos < N; pos += blockDim.x)
    {
        s_data[pos] = d_Src[pos];
    }

    //Main radix-4 stages
    const int pos = threadIdx.x;

    for (int stride = N >> 2; stride > 0; stride >>= 2)
    {
        int lo = pos & (stride - 1);
        int i0 = ((pos - lo) << 2) + lo;
        int i1 = i0 + stride;
        int i2 = i1 + stride;
        int i3 = i2 + stride;

        __syncthreads();
        scalar_t D0 = s_data[i0];
        scalar_t D1 = s_data[i1];
        scalar_t D2 = s_data[i2];
        scalar_t D3 = s_data[i3];

        scalar_t T;
        T = D0;
        D0         = D0 + D2;
        D2         = T - D2;
        T = D1;
        D1         = D1 + D3;
        D3         = T - D3;
        T = D0;
        s_data[i0] = D0 + D1;
        s_data[i1] = T - D1;
        T = D2;
        s_data[i2] = D2 + D3;
        s_data[i3] = T - D3;
    }

    //Do single radix-2 stage for odd power of two
    if (log2d & 1)
    {
        __syncthreads();

        for (int pos = threadIdx.x; pos < N / 2; pos += blockDim.x)
        {
            int i0 = pos << 1;
            int i1 = i0 + 1;

            scalar_t D0 = s_data[i0];
            scalar_t D1 = s_data[i1];
            s_data[i0] = D0 + D1;
            s_data[i1] = D0 - D1;
        }
    }

    __syncthreads();

    for (int pos = threadIdx.x; pos < N; pos += blockDim.x)
    {
        d_Dst[pos] = s_data[pos];
    }
}


/*
CPU front-end for batched FWHT on tensor X.
Creates the grid and launches kernels.

:param X: input tensor of shape (batch_size, D). This tensor is overwritten.
:return: overwritten tensor X with the result of batched FWHT.
*/
at::Tensor fwht_cuda_frontend(at::Tensor X) {
	const int num_threads = 256;  // This can be changed to a different power of two.

	auto shape = X.sizes();
	int batch_size = shape[0];
	int log2D = (int) log2((float) shape[1]);

	int D = 1 << log2D;
	int blocks_per_grid = D / (4 * num_threads);
	int threads_per_block = batch_size;
	dim3 grid(blocks_per_grid, threads_per_block, 1);  // Create a 3D grid.

	// Launch kernels
	for (; log2D > ELEMENTARY_LOG2SIZE; log2D -= 2, D >>= 2, batch_size <<= 2) {
		AT_DISPATCH_FLOATING_TYPES(X.type(), "fwht_batch2_kernel", ([&] {
			fwht_batch2_kernel<<<grid, num_threads>>>(X.data<scalar_t>(), D / 4);
		}));
	}

	// Launch kernel
	AT_DISPATCH_FLOATING_TYPES(X.type(), "fwht_batch2_kernel", ([&] {
		fwht_batch1_kernel<<<batch_size, D / 4, D * sizeof(scalar_t)>>>(X.data<scalar_t>(), log2D);
	}));

	return X;
}
